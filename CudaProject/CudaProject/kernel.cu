#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "CImg.h"

using namespace cimg_library;

__global__ void gpu_taskR(unsigned char * d_in, unsigned char * d_out, int img_width, int img_height) {

	unsigned long scale = img_height * img_width;
	unsigned long r = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned long g = r + scale;
	unsigned long b = g + scale;



	d_out[r] = d_in[r];
	d_out[g] = 0;
	d_out[b] = 0;

}

__global__ void gpu_taskG(unsigned char * d_in, unsigned char * d_out1, int img_width, int img_height) {

	unsigned long scale = img_height * img_width;
	unsigned long r = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned long g = r + scale;
	unsigned long b = g + scale;

	d_out1[r] = 0;
	d_out1[g] = d_in[g];
	d_out1[b] = 0;

}

__global__ void gpu_taskB(unsigned char * d_in, unsigned char * d_out2, int img_width, int img_height) {

	unsigned long scale = img_height * img_width;
	unsigned long r = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned long g = r + scale;
	unsigned long b = g + scale;

	d_out2[r] = d_in[r];
	d_out2[g] = d_in[g];
	d_out2[b] = d_in[b];

}

int main(int argc, char ** argv) {

	clock_t tinicio, tfim, tdecorrido;

	tinicio = clock();

	CImg<unsigned char> img("imagem2.bmp");
	unsigned long img_size = img.size();
	int img_width = img.width();
	int img_height = img.height();
	int img_depth = img.depth();
	int img_dim = img.spectrum();
	unsigned char *h_in = img.data();

	CImg<unsigned char> img_out(img_width, img_height, img_depth, img_dim);
	CImg<unsigned char> img_out1(img_width, img_height, img_depth, img_dim);
	CImg<unsigned char> img_out2(img_width, img_height, img_depth, img_dim);
	unsigned char *h_out = img_out.data();

	//declara ponteiros para mem�ria da gpu
	unsigned char * d_in;
	unsigned char * d_out;
	
	//aloca mem�ria na gpu
	hipMalloc((void**)&d_in, img_size);
	hipMalloc((void**)&d_out, img_size);

	//transfere dados para a gpu
	hipMemcpy(d_in, h_in, img_size, hipMemcpyHostToDevice);

	dim3 grid(512);         // 512 x 1 x 1
	dim3 block(1024); // 1024 x 1024 x 1

	//executa comando kernel
	gpu_taskR <<<grid, block >>> (d_in, d_out, img_width, img_height);

	//espera a execu��o da gpu
	hipDeviceSynchronize();

	//transfere dados para a cpu devolta
	hipMemcpy(d_in, h_in, img_size, hipMemcpyDeviceToHost);

	gpu_taskG << <grid, block >> > (d_in, d_out, img_width, img_height);
	hipDeviceSynchronize();

	gpu_taskB << <grid, block >> > (d_in, d_out, img_width, img_height);
	hipDeviceSynchronize();


	for (int i=0; i < img_width; i++) {
		for (int j = 0; j < img_height; j++) {
			img_out(i, j, 0) = img(i, j, 0);
			img_out(i, j, 1) = 0;
			img_out(i, j, 2) = 0;
		}	
	}

	for (int i = 0; i < img_width; i++) {
		for (int j = 0; j < img_height; j++) {
			img_out1(i, j, 1) = img(i, j, 0);
			img_out1(i, j, 2) = 0;
			img_out1(i, j, 0) = 0;
		}
	}

	for (int i = 0; i < img_width; i++) {
		for (int j = 0; j < img_height; j++) {
			img_out2(i, j, 2) = img(i, j, 0);
			img_out2(i, j, 1) = 0;
			img_out2(i, j, 0) = 0;
		}
	}
	
	//CImgDisplay main_disp(img_out, "Ap�s Processar RED");
	//CImgDisplay main_disp1(img_out1, "Ap�s Processar GREEN");
	//CImgDisplay main_disp2(img_out2, "Ap�s Processar BLUE");
	
	img_out.save("RED.bmp", -1, 3);
	img_out1.save("GREEN.bmp", -1, 5);
	img_out2.save("BLUE.bmp", -1, 4);
	/*
	while (!main_disp.is_closed()) {
		main_disp.wait();
	}

	while (!main_disp1.is_closed()) {
		main_disp.wait();
	}

	while (!main_disp2.is_closed()) {
		main_disp.wait();
	}
	*/

	tfim = clock();
	tdecorrido = ((tfim - tinicio) / (CLOCKS_PER_SEC / 1000));

	hipFree(d_in);
	hipFree(d_out);

	printf("TEMPO: %d milseg \n", tdecorrido);

	return 0;
}



